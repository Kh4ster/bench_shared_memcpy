#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/device_buffer.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <hip/hip_runtime_api.h>

using namespace cooperative_groups;

// Dummy compute just for the sake of having one
template <typename T>
__device__
static void compute(thread_group g, T shared[])
{
    shared[g.thread_rank()] += shared[g.thread_rank() + g.size()];
}

template <typename T, int block_width>
__global__
static void basic(cuda_tools::device_buffer<T> result,
                  cuda_tools::device_buffer<T> global1,
                  cuda_tools::device_buffer<T> global2)
{
    __shared__ T shared[block_width * 2];
    auto group = cooperative_groups::this_thread_block();

    const int tx = threadIdx.x;
    const int gx = tx + blockIdx.x * blockDim.x;
    if (gx >= result.size_) return;
 
    shared[group.thread_rank()               ] = global1[gx];
    shared[group.size() + group.thread_rank()] = global2[gx];
 
    group.sync(); // Wait for all copies to complete
 
    compute(group, shared);

    result[gx] = shared[group.thread_rank()];
}

template <typename T, int block_width>
__global__
static void cooperative_basic(cuda_tools::device_buffer<T> result,
                              cuda_tools::device_buffer<T> global1,
                              cuda_tools::device_buffer<T> global2,
                              const int subset_count)
{
    __shared__ T shared[block_width * 2];
    auto group = cooperative_groups::this_thread_block();

    const int tx = threadIdx.x;
    const int gx = tx + blockIdx.x * blockDim.x;
    if (gx * subset_count >= result.size_) return;
    const int grid_size = blockDim.x * gridDim.x;
 
    for (int subset = 0; subset < subset_count; ++subset)
    {
        shared[group.thread_rank()               ] = global1[subset * grid_size + gx];
        shared[group.size() + group.thread_rank()] = global2[subset * grid_size + gx];
 
        group.sync(); // Wait for all copies to complete
 
        compute(group, shared);

        result[gx + subset * grid_size] = shared[group.thread_rank()];

        group.sync();
    }
}

template <typename T, int block_width>
__global__
static void cooperative_async(cuda_tools::device_buffer<T> result,
                              cuda_tools::device_buffer<T> global1,
                              cuda_tools::device_buffer<T> global2,
                              const int subset_count)
{
    __shared__ T shared[block_width * 2];
    auto group = cooperative_groups::this_thread_block();

    const int tx = threadIdx.x;
    const int gx = tx + blockIdx.x * blockDim.x;
    if (gx * subset_count >= result.size_) return;
    const int grid_size = blockDim.x * gridDim.x;
 
    for (int subset = 0; subset < subset_count; ++subset)
    {
        cooperative_groups::memcpy_async(group, shared,
            &global1[subset * grid_size + gx], sizeof(T) * group.size());
        cooperative_groups::memcpy_async(group, shared + group.size(),
            &global2[subset * grid_size + gx], sizeof(T) * group.size());

        cooperative_groups::wait(group); // Wait for all copies to complete
 
        compute(group, shared);

        result[gx + subset * grid_size] = shared[group.thread_rank()];

        group.sync();
    }
}

void basic(cuda_tools::host_shared_ptr<int> _result,
           cuda_tools::host_shared_ptr<int> _global1,
           cuda_tools::host_shared_ptr<int> _global2)
{
    constexpr int TILE_WIDTH  = 64;
    constexpr int TILE_HEIGHT = 1;

    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(basic<int), TILE_WIDTH>, hipFuncCachePreferShared);
    
    cuda_tools::device_buffer<int> result(_result);
    cuda_tools::device_buffer<int> global1(_global1);
    cuda_tools::device_buffer<int> global2(_global2);

    const int gx             = (result.size_ + TILE_WIDTH - 1) / (TILE_WIDTH);
    const int gy             = 1;

    const dim3 block(TILE_WIDTH, TILE_HEIGHT);
    const dim3 grid(gx, gy);

    basic<int, TILE_WIDTH><<<grid, block>>>(result, global1, global2);
    kernel_check_error();

    hipDeviceSynchronize();
    hipProfilerStop();
}

void cooperative_basic(cuda_tools::host_shared_ptr<int> _result,
                       cuda_tools::host_shared_ptr<int> _global1,
                       cuda_tools::host_shared_ptr<int> _global2)
{
    constexpr int TILE_WIDTH  = 64;
    constexpr int TILE_HEIGHT = 1;
    constexpr int SUBSET_COUNT = 4;

    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cooperative_basic<int), TILE_WIDTH>, hipFuncCachePreferShared);

    cuda_tools::device_buffer<int> result(_result);
    cuda_tools::device_buffer<int> global1(_global1);
    cuda_tools::device_buffer<int> global2(_global2);

    const int gx             = (result.size_ + TILE_WIDTH - 1) / (TILE_WIDTH * SUBSET_COUNT);
    const int gy             = 1;
    
    const dim3 block(TILE_WIDTH, TILE_HEIGHT);
    const dim3 grid(gx, gy);

    cooperative_basic<int, TILE_WIDTH><<<grid, block>>>(result, global1, global2, SUBSET_COUNT);
    kernel_check_error();

    hipDeviceSynchronize();
    hipProfilerStop();
}

void cooperative_async(cuda_tools::host_shared_ptr<int> _result,
    cuda_tools::host_shared_ptr<int> _global1,
    cuda_tools::host_shared_ptr<int> _global2)
{
    constexpr int TILE_WIDTH  = 64;
    constexpr int TILE_HEIGHT = 1;
    constexpr int SUBSET_COUNT = 1;

    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cooperative_async<int), TILE_WIDTH>, hipFuncCachePreferShared);

    cuda_tools::device_buffer<int> result(_result);
    cuda_tools::device_buffer<int> global1(_global1);
    cuda_tools::device_buffer<int> global2(_global2);

    const int gx             = (result.size_ + TILE_WIDTH - 1) / (TILE_WIDTH * SUBSET_COUNT);
    const int gy             = 1;

    const dim3 block(TILE_WIDTH, TILE_HEIGHT);
    const dim3 grid(gx, gy);

    cooperative_async<int, TILE_WIDTH><<<grid, block>>>(result, global1, global2, SUBSET_COUNT);
    kernel_check_error();

    hipDeviceSynchronize();
    hipProfilerStop();
}